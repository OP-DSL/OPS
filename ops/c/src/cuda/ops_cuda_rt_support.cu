#include "hip/hip_runtime.h"
/*
* Open source copyright declaration based on BSD open source template:
* http://www.opensource.org/licenses/bsd-license.php
*
* This file is part of the OPS distribution.
*
* Copyright (c) 2013, Mike Giles and others. Please see the AUTHORS file in
* the main source directory for a full list of copyright holders.
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
* * Redistributions of source code must retain the above copyright
* notice, this list of conditions and the following disclaimer.
* * Redistributions in binary form must reproduce the above copyright
* notice, this list of conditions and the following disclaimer in the
* documentation and/or other materials provided with the distribution.
* * The name of Mike Giles may not be used to endorse or promote products
* derived from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY Mike Giles ''AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
* WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL Mike Giles BE LIABLE FOR ANY
* DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
* (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
* LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
* ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

/** @file
  * @brief OPS cuda specific runtime support functions
  * @author Gihan Mudalige
  * @details Implements cuda backend runtime support functions
  */

//
// header files
//

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_math_constants.h>

#include <ops_cuda_rt_support.h>
#include <ops_lib_core.h>

/*__global__ void copy_kernel(char *dest, char *src, int size ) {
  int tid = blockIdx.x;
  memcpy(&dest[tid],&src[tid],size);
}*/

__global__ void copy_kernel_tobuf(char *dest, char *src, int rx_s, int rx_e,
                                  int ry_s, int ry_e, int rz_s, int rz_e,
                                  int x_step, int y_step, int z_step,
                                  int size_x, int size_y, int size_z,
                                  int buf_strides_x, int buf_strides_y,
                                  int buf_strides_z, int type_size, int dim, int OPS_soa) {

  int idx_z = rz_s + z_step * (blockDim.z * blockIdx.z + threadIdx.z);
  int idx_y = ry_s + y_step * (blockDim.y * blockIdx.y + threadIdx.y);
  int idx_x = rx_s + x_step * (blockDim.x * blockIdx.x + threadIdx.x);

  if ((x_step == 1 ? idx_x < rx_e : idx_x > rx_e) &&
      (y_step == 1 ? idx_y < ry_e : idx_y > ry_e) &&
      (z_step == 1 ? idx_z < rz_e : idx_z > rz_e)) {

    if (OPS_soa) src += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * type_size;
    else src += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * type_size * dim;
    dest += ((idx_z - rz_s) * z_step * buf_strides_z +
             (idx_y - ry_s) * y_step * buf_strides_y +
             (idx_x - rx_s) * x_step * buf_strides_x) *
            type_size * dim;
    for (int d = 0; d < dim; d++) {
      memcpy(dest+d*type_size, src, type_size);
      if (OPS_soa) src += size_x * size_y * size_z * type_size;
      else src += type_size;
    }
  }
}

__global__ void copy_kernel_frombuf(char *dest, char *src, int rx_s, int rx_e,
                                    int ry_s, int ry_e, int rz_s, int rz_e,
                                    int x_step, int y_step, int z_step,
                                    int size_x, int size_y, int size_z,
                                    int buf_strides_x, int buf_strides_y,
                                    int buf_strides_z, int type_size, int dim, int OPS_soa) {

  int idx_z = rz_s + z_step * (blockDim.z * blockIdx.z + threadIdx.z);
  int idx_y = ry_s + y_step * (blockDim.y * blockIdx.y + threadIdx.y);
  int idx_x = rx_s + x_step * (blockDim.x * blockIdx.x + threadIdx.x);

  if ((x_step == 1 ? idx_x < rx_e : idx_x > rx_e) &&
      (y_step == 1 ? idx_y < ry_e : idx_y > ry_e) &&
      (z_step == 1 ? idx_z < rz_e : idx_z > rz_e)) {

    if (OPS_soa) dest += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * type_size;
    else dest += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * type_size * dim;
    src += ((idx_z - rz_s) * z_step * buf_strides_z +
            (idx_y - ry_s) * y_step * buf_strides_y +
            (idx_x - rx_s) * x_step * buf_strides_x) *
           type_size * dim;
    for (int d = 0; d < dim; d++) {
      memcpy(dest, src + d*type_size, type_size);
      if (OPS_soa) dest += size_x * size_y * size_z * type_size;
      else dest += type_size;
    }
  }
}

void ops_halo_copy_tobuf(char *dest, int dest_offset, ops_dat src, int rx_s,
                         int rx_e, int ry_s, int ry_e, int rz_s, int rz_e,
                         int x_step, int y_step, int z_step, int buf_strides_x,
                         int buf_strides_y, int buf_strides_z) {

  dest += dest_offset;
  int thr_x = abs(rx_s - rx_e);
  int blk_x = 1;
  if (abs(rx_s - rx_e) > 8) {
    blk_x = (thr_x - 1) / 8 + 1;
    thr_x = 8;
  }
  int thr_y = abs(ry_s - ry_e);
  int blk_y = 1;
  if (abs(ry_s - ry_e) > 8) {
    blk_y = (thr_y - 1) / 8 + 1;
    thr_y = 8;
  }
  int thr_z = abs(rz_s - rz_e);
  int blk_z = 1;
  if (abs(rz_s - rz_e) > 8) {
    blk_z = (thr_z - 1) / 8 + 1;
    thr_z = 8;
  }

  dim3 grid(blk_x, blk_y, blk_z);
  dim3 tblock(thr_x, thr_y, thr_z);
  copy_kernel_tobuf<<<grid, tblock>>>(
      dest, src->data_d, rx_s, rx_e, ry_s, ry_e, rz_s, rz_e, x_step, y_step,
      z_step, src->size[0], src->size[1], src->size[2], buf_strides_x,
      buf_strides_y, buf_strides_z, src->type_size, src->dim, src->block->instance->OPS_soa);
  cutilSafeCall(src->block->instance->ostream(),hipGetLastError());

  // TODO: MPI buffers and GPUDirect
}

void ops_halo_copy_frombuf(ops_dat dest, char *src, int src_offset, int rx_s,
                           int rx_e, int ry_s, int ry_e, int rz_s, int rz_e,
                           int x_step, int y_step, int z_step,
                           int buf_strides_x, int buf_strides_y,
                           int buf_strides_z) {

  src += src_offset;
  int thr_x = abs(rx_s - rx_e);
  int blk_x = 1;
  if (abs(rx_s - rx_e) > 8) {
    blk_x = (thr_x - 1) / 8 + 1;
    thr_x = 8;
  }
  int thr_y = abs(ry_s - ry_e);
  int blk_y = 1;
  if (abs(ry_s - ry_e) > 8) {
    blk_y = (thr_y - 1) / 8 + 1;
    thr_y = 8;
  }
  int thr_z = abs(rz_s - rz_e);
  int blk_z = 1;
  if (abs(rz_s - rz_e) > 8) {
    blk_z = (thr_z - 1) / 8 + 1;
    thr_z = 8;
  }

  dim3 grid(blk_x, blk_y, blk_z);
  dim3 tblock(thr_x, thr_y, thr_z);
  copy_kernel_frombuf<<<grid, tblock>>>(
      dest->data_d, src, rx_s, rx_e, ry_s, ry_e, rz_s, rz_e, x_step, y_step,
      z_step, dest->size[0], dest->size[1], dest->size[2], buf_strides_x,
      buf_strides_y, buf_strides_z, dest->type_size, dest->dim, dest->block->instance->OPS_soa);
  cutilSafeCall(dest->block->instance->ostream(),hipGetLastError());
  dest->dirty_hd = 2;
}

template <int dir>
__global__ void ops_internal_copy_cuda_kernel(char * dat0_p, char *dat1_p,
         int s0, int s01, int start0, int end0,
#if OPS_MAX_DIM>1
        int s1, int s11, int start1, int end1,
#if OPS_MAX_DIM>2
        int s2, int s21, int start2, int end2,
#if OPS_MAX_DIM>3
        int s3, int s31, int start3, int end3,
#if OPS_MAX_DIM>4
        int s4, int s41, int start4, int end4,
#endif
#endif
#endif
#endif
        int dim, int type_size,
        int OPS_soa) {
  int i = start0 + threadIdx.x + blockIdx.x*blockDim.x;
  int j = start1 + threadIdx.y + blockIdx.y*blockDim.y;
  int rest = threadIdx.z + blockIdx.z*blockDim.z;
  int mult = OPS_soa ? type_size : dim*type_size;

    long fullsize = s0;
    long fullsize1 = s01;
    long idx = i*mult;
    long idx1 = i*mult;
#if OPS_MAX_DIM>1
    fullsize *= s1;
    fullsize1 *= s11;
    idx += j * s0 * mult;
    idx1 += j * s01 * mult;
#endif
#if OPS_MAX_DIM>2
    fullsize *= s2;
    fullsize1 *= s21;
    const int sz2 = end2-start2;
    int nextSize = rest / sz2;
    int k = start2+rest - nextSize*sz2;
    rest = nextSize;
    idx += k * s0 * s1 * mult;
    idx1 += k * s01 * s11 * mult;
#endif
#if OPS_MAX_DIM>3
    fullsize *= s3;
    fullsize1 *= s31;
    const int sz3 = end3-start3;
    nextSize = rest / sz3;
    int l = start3 + rest - nextSize*sz3;
    rest = nextSize;
    idx += l * s0 * s1 * s2 * mult;
    idx1 += l * s01 * s11 * s21 * mult;
#endif
#if OPS_MAX_DIM>4
    fullsize *= s4;
    fullsize1 *= s41;
    const int sz4 = end4-start4;
    nextSize = rest / sz4;
    int m = start4 + rest - nextSize*sz4;
    idx += m * s0 * s1 * s2 * s3 * mult;
    idx1 += m * s01 * s11 * s21 * s31 * mult;
#endif
    if (i<end0
#if OPS_MAX_DIM>1
        && j < end1
#if OPS_MAX_DIM>2
        && k < end2
#if OPS_MAX_DIM>3
        && l < end3
#if OPS_MAX_DIM>4
        && m < end4
#endif
#endif
#endif
#endif
       )

    if (OPS_soa) {
      for (int d = 0; d < dim; d++)
        for (int c = 0; c < type_size; c++)
          if (dir == 0)
            dat1_p[idx1+d*fullsize1*type_size+c] = dat0_p[idx+d*fullsize*type_size+c];
          else
            dat0_p[idx+d*fullsize*type_size+c] = dat1_p[idx1+d*fullsize1*type_size+c];
    } else
      for (int d = 0; d < dim*type_size; d++)
        if (dir == 0)
          dat1_p[idx1+d] = dat0_p[idx+d];
        else
          dat0_p[idx+d] = dat1_p[idx1+d];

}


void ops_internal_copy_cuda(ops_kernel_descriptor *desc) {
  int reverse = strcmp(desc->name, "ops_internal_copy_cuda_reverse")==0;
  int range[2*OPS_MAX_DIM]={0};
  for (int d = 0; d < desc->dim; d++) {
    range[2*d] = desc->range[2*d];
    range[2*d+1] = desc->range[2*d+1];
  }
  for (int d = desc->dim; d < OPS_MAX_DIM; d++) {
    range[2*d] = 0;
    range[2*d+1] = 1;
  }
  ops_dat dat0 = desc->args[0].dat;
  ops_dat dat1 = desc->args[1].dat;
  double __t1=0.0,__t2=0.0,__c1,__c2;
  if (dat0->block->instance->OPS_diags>1) {
    dat0->block->instance->OPS_kernels[-1].count++;
    ops_timers_core(&__c1,&__t1);
  }
  char *dat0_p = desc->args[0].data_d + desc->args[0].dat->base_offset;
  char *dat1_p = desc->args[1].data_d + desc->args[1].dat->base_offset;
  int s0 = dat0->size[0];
  int s01 = dat1->size[0];
#if OPS_MAX_DIM>1
  int s1 = dat0->size[1];
  int s11 = dat1->size[1];
#if OPS_MAX_DIM>2
  int s2 = dat0->size[2];
  int s21 = dat1->size[2];
#if OPS_MAX_DIM>3
  int s3 = dat0->size[3];
  int s31 = dat1->size[3];
#if OPS_MAX_DIM>4
  int s4 = dat0->size[4];
  int s41 = dat1->size[4];
#endif
#endif
#endif
#endif
  
  dim3 grid((range[2*0+1]-range[2*0] - 1) / dat0->block->instance->OPS_block_size_x + 1,
            (range[2*1+1]-range[2*1] - 1) / dat0->block->instance->OPS_block_size_y + 1,
           ((range[2*2+1]-range[2*2] - 1) / dat0->block->instance->OPS_block_size_z + 1) *
            (range[2*3+1]-range[2*3]) *
            (range[2*4+1]-range[2*4]));
  dim3 tblock(dat0->block->instance->OPS_block_size_x,
              dat0->block->instance->OPS_block_size_y,
              dat0->block->instance->OPS_block_size_z);

  if (grid.x>0 && grid.y>0 && grid.z>0) {
    if (reverse)
    ops_internal_copy_cuda_kernel<1><<<grid,tblock>>>(
        dat0_p,
        dat1_p,
        s0,s01, range[2*0], range[2*0+1],
#if OPS_MAX_DIM>1
        s1,s11, range[2*1], range[2*1+1],
#if OPS_MAX_DIM>2
        s2,s21, range[2*2], range[2*2+1],
#if OPS_MAX_DIM>3
        s3,s31, range[2*3], range[2*3+1],
#if OPS_MAX_DIM>4
        s4,s41, range[2*4], range[2*4+1],
#endif
#endif
#endif
#endif
        dat0->dim, dat0->type_size,
        dat0->block->instance->OPS_soa
        );
    else
    ops_internal_copy_cuda_kernel<0><<<grid,tblock>>>(
        dat0_p,
        dat1_p,
        s0,s01, range[2*0], range[2*0+1],
#if OPS_MAX_DIM>1
        s1,s11, range[2*1], range[2*1+1],
#if OPS_MAX_DIM>2
        s2,s21, range[2*2], range[2*2+1],
#if OPS_MAX_DIM>3
        s3,s31, range[2*3], range[2*3+1],
#if OPS_MAX_DIM>4
        s4,s41, range[2*4], range[2*4+1],
#endif
#endif
#endif
#endif
        dat0->dim, dat0->type_size,
        dat0->block->instance->OPS_soa
        );
    cutilSafeCall(dat0->block->instance->ostream(),hipGetLastError());
  }
  if (dat0->block->instance->OPS_diags>1) {
    cutilSafeCall(dat0->block->instance->ostream(), hipDeviceSynchronize());
    ops_timers_core(&__c2,&__t2);
    int start[OPS_MAX_DIM];
    int end[OPS_MAX_DIM];
    for ( int n=0; n<desc->dim; n++ ){
      start[n] = range[2*n];end[n] = range[2*n+1];
    }
    dat0->block->instance->OPS_kernels[-1].time += __t2-__t1;
    dat0->block->instance->OPS_kernels[-1].transfer += ops_compute_transfer(desc->dim, start, end, &desc->args[0]);
    dat0->block->instance->OPS_kernels[-1].transfer += ops_compute_transfer(desc->dim, start, end, &desc->args[1]);
  }
}
